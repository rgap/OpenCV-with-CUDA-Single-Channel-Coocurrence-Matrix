#include "hip/hip_runtime.h"

#include "SCM_GPU.cuh"

#include <iostream>
#include <fstream>
#include <math.h>


#include <hip/hip_runtime.h>
#include <>

#include "cuPrintf.cu"

#if __CUDA_ARCH__ < 200 	//Compute capability 1.x architectures
#define CUPRINTF cuPrintf 
#else						//Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
								blockIdx.y*gridDim.x+blockIdx.x,\
								threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
								__VA_ARGS__)
#endif


SCM_GPU::SCM_GPU(Image *imagen, int numNiveles){

	this->imagen = imagen;
	this->pixelMat_h = imagen->pixelMat;

	this->numNiveles = numNiveles;

	rows = numNiveles;
    cols = numNiveles;

	arraySize = rows*cols;
	
	//AsignarMemoria
    matrizSCM_h = new int[arraySize];

    //Inicializar con 0
    for (int i = 0; i < arraySize; ++i)
		matrizSCM_h[i] = 0;

	//Inicializar memoria device
	inicializar_MemDevice();

	//Inicializar variables GPU
	inicializar_VarsGPU();

}

void SCM_GPU::inicializar_MemDevice(){

	//////////// Memoria de imagen en device //////////////////

	size_t size_imagen = imagen->arraySize*sizeof(Pixel);

	hipMalloc((void**)&pixelMat_d, size_imagen);

	hipMemcpy(pixelMat_d,pixelMat_h,size_imagen,hipMemcpyHostToDevice);


	//////////// Memoria de SCM en device //////////////////

	size_t size_matrizSCM = arraySize*sizeof(int);

	hipMalloc((void**)&matrizSCM_d, size_matrizSCM);

	hipMemcpy(matrizSCM_d,matrizSCM_h,size_matrizSCM,hipMemcpyHostToDevice);
}

void SCM_GPU::inicializar_VarsGPU(){

	int numThreads = 32;

	Grid_Dim_x=imagen->rows, Grid_Dim_y=imagen->cols;	//Grid structure values

	Block_Dim_x=numThreads, Block_Dim_y=numThreads;	//Block structure values


	Grid = dim3( ceil(Grid_Dim_x*1.0/numThreads), ceil(Grid_Dim_y*1.0/numThreads) );		//Grid structure

	Block = dim3(Block_Dim_x,Block_Dim_x);	//Block structure, threads/block limited by specific device
}


__global__ 
void createSCM_kernel(Pixel*pixelMat,int imgRows,int imgCols,
					  int*matrizSCM, int numNiveles, int delta_i,int delta_j)
					  
{

	int i = blockIdx.y*blockDim.y+threadIdx.y;
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	
    int cols = numNiveles;

	int posx,posy;

	//CUPRINTF("i = %d  \n",i);
	//CUPRINTF("j = %d  \n",j);

    if ( (j + delta_j) < imgCols && (i + delta_i) < imgRows && 
		((j + delta_j) >= 0) && ((i + delta_i) >= 0) )
	{
        posx = pixelMat[i*imgCols+j].canal_0;
		posy = pixelMat[(i + delta_i)*imgCols+(j + delta_j)].canal_0;

		atomicAdd(&matrizSCM[posx*cols+posy], 1);
		
        atomicAdd(&matrizSCM[posy*cols+posx], 1);

    }
	__syncthreads();
}


void SCM_GPU::createSCM(int* offsets, int distance, int direction){

	this->distance = distance;
	this->direction = direction;
	
	this->delta_j = offsets[(this->direction)*this->distance];
    this->delta_i = offsets[(this->direction + 1)*this->distance];
	
	int posx,posy,i,j;

	//Inicializar con 0
    for (int i = 0; i < arraySize; ++i)
		matrizSCM_h[i] = 0;


	//Ejecutar kernel
	createSCM_kernel<<<Grid,Block>>>(pixelMat_d,imagen->rows,imagen->cols,
									 matrizSCM_d,numNiveles,delta_i,delta_j);

	hipDeviceSynchronize();


	copiar_DeviceHost();
}


void SCM_GPU::copiar_DeviceHost(){

	/////////////////// Variables del device al host ///////////////////////// 

	// Retrieve result from device to host in the host memory
	//hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
	hipMemcpy(matrizSCM_h,matrizSCM_d,sizeof(int)*arraySize,hipMemcpyDeviceToHost);
}


void SCM_GPU::showFileSCM(){

	ofstream ficheroSCM("SCM_GPU.txt");

	for (int i = 0; i < arraySize; ++i) {
		if(i%cols==0) ficheroSCM<<endl;

		ficheroSCM<<matrizSCM_h[i]<<'\t';
		
	}
	//cout<<"TERMINO\n";
	ficheroSCM.close();
}


SCM_GPU::~SCM_GPU() {

	// liberar memoria en el device y host 
	//free(imagen_h);
	//hipFree(imagen_d);
	free(matrizSCM_h);
	hipFree(matrizSCM_d);
}